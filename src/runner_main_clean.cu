#include "hip/hip_runtime.h"
/*******************************************************************************
 * This file is part of SWIFT.
 * Copyright (c) 2012 Pedro Gonnet (pedro.gonnet@durham.ac.uk)
 *                    Matthieu Schaller (schaller@strw.leidenuniv.nl)
 *               2015 Peter W. Draper (p.w.draper@durham.ac.uk)
 *               2022 Abouzied M. A. Nasar (abouzied.nasar@manchester.ac.uk)
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ******************************************************************************/
/* Config parameters. */
#define GPUOFFLOAD_DENSITY 1   // off-load hydro density to GPU
#define GPUOFFLOAD_GRADIENT 1  // off-load hydro gradient to GPU
#define GPUOFFLOAD_FORCE 1     // off-load hydro force to GPU

//A. Nasar: Remove as will no longer be necessary. Leaving for now during dev
#define RECURSE 1 //Allow recursion through sub-tasks before offloading

#include "../config.h"

/* MPI headers. */
#ifdef WITH_MPI
#include <mpi.h>
#endif

#ifdef __cplusplus
extern "C" {
#endif

/* Config parameters. */
#include <config.h>

/* MPI headers. */
#ifdef WITH_MPI
#include <mpi.h>
#endif

/* This object's header. */
#include "runner.h"

/* Local headers. */
#include "engine.h"
#include "feedback.h"
#include "scheduler.h"
#include "space_getsid.h"
#include "timers.h"

/* Import the gravity loop functions. */
#include "runner_doiact_grav.h"

/* Import the density loop functions. */
#define FUNCTION density
#define FUNCTION_TASK_LOOP TASK_LOOP_DENSITY
#include "runner_doiact_hydro.h"
#include "runner_doiact_undef.h"

/* Import the gradient loop functions (if required). */
#ifdef EXTRA_HYDRO_LOOP
#define FUNCTION gradient
#define FUNCTION_TASK_LOOP TASK_LOOP_GRADIENT
#include "runner_doiact_hydro.h"
#include "runner_doiact_undef.h"
#endif

/* Import the force loop functions. */
#define FUNCTION force
#define FUNCTION_TASK_LOOP TASK_LOOP_FORCE
#include "runner_doiact_hydro.h"
#include "runner_doiact_undef.h"

/* Import the limiter loop functions. */
#define FUNCTION limiter
#define FUNCTION_TASK_LOOP TASK_LOOP_LIMITER
#include "runner_doiact_limiter.h"
#include "runner_doiact_undef.h"

/* Import the stars density loop functions. */
#define FUNCTION density
#define FUNCTION_TASK_LOOP TASK_LOOP_DENSITY
#include "runner_doiact_stars.h"
#include "runner_doiact_undef.h"

#ifdef EXTRA_STAR_LOOPS

/* Import the stars prepare1 loop functions. */
#define FUNCTION prep1
#define FUNCTION_TASK_LOOP TASK_LOOP_STARS_PREP1
#include "runner_doiact_stars.h"
#include "runner_doiact_undef.h"

/* Import the stars prepare2 loop functions. */
#define FUNCTION prep2
#define FUNCTION_TASK_LOOP TASK_LOOP_STARS_PREP2
#include "runner_doiact_stars.h"
#include "runner_doiact_undef.h"

#endif /* EXTRA_STAR_LOOPS */

/* Import the stars feedback loop functions. */
#define FUNCTION feedback
#define FUNCTION_TASK_LOOP TASK_LOOP_FEEDBACK
#include "runner_doiact_stars.h"
#include "runner_doiact_undef.h"

/* Import the black hole density loop functions. */
#define FUNCTION density
#define FUNCTION_TASK_LOOP TASK_LOOP_DENSITY
#include "runner_doiact_black_holes.h"
#include "runner_doiact_undef.h"

/* Import the black hole feedback loop functions. */
#define FUNCTION swallow
#define FUNCTION_TASK_LOOP TASK_LOOP_SWALLOW
#include "runner_doiact_black_holes.h"
#include "runner_doiact_undef.h"

/* Import the black hole feedback loop functions. */
#define FUNCTION feedback
#define FUNCTION_TASK_LOOP TASK_LOOP_FEEDBACK
#include "runner_doiact_black_holes.h"
#include "runner_doiact_undef.h"

/* Import the sink density loop functions. */
#define FUNCTION density
#define FUNCTION_TASK_LOOP TASK_LOOP_DENSITY
#include "runner_doiact_sinks.h"
#include "runner_doiact_undef.h"

/* Import the sink swallow loop functions. */
#define FUNCTION swallow
#define FUNCTION_TASK_LOOP TASK_LOOP_SWALLOW
#include "runner_doiact_sinks.h"
#include "runner_doiact_undef.h"

/* Import the RT gradient loop functions */
#define FUNCTION rt_gradient
#define FUNCTION_TASK_LOOP TASK_LOOP_RT_GRADIENT
#include "runner_doiact_hydro.h"
#include "runner_doiact_undef.h"

/* Import the RT transport (force) loop functions. */
#define FUNCTION rt_transport
#define FUNCTION_TASK_LOOP TASK_LOOP_RT_TRANSPORT
#include "runner_doiact_hydro.h"
#include "runner_doiact_undef.h"

#ifdef __cplusplus
}
#endif
/**
 * @brief The #runner main thread routine.
 *
 * @param data A pointer to this thread's data.
 **/

/* CUDA Header. Wrap in extern "C" to prevent C++ function name mangling */
#ifdef WITH_CUDA
#ifdef __cplusplus
extern "C" {
#endif

#include "cuda/part_gpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "runner_doiact_functions_hydro_gpu.h"
#include "runner_gpu_pack_functions.h"
#include "cuda/GPU_runner_functions.h"

#ifdef __cplusplus
}
#endif

void *runner_main2(void *data) {
  struct runner *r = (struct runner *)data;
  struct engine *e = r->e;
  struct scheduler *sched = &e->sched;
  struct space *space = e->s;

  //////////Declare and allocate GPU launch control data structures/////////
  /*pack_vars contain data required for self and pair packing tasks destined
   *  for the GPU*/
  //A. N: Needed
  struct pack_vars_self *pack_vars_self_dens;
  struct pack_vars_self *pack_vars_self_forc;
  struct pack_vars_self *pack_vars_self_grad;
  struct pack_vars_pair *pack_vars_pair_dens;
  struct pack_vars_pair *pack_vars_pair_forc;
  struct pack_vars_pair *pack_vars_pair_grad;

  hipHostMalloc((void **)&pack_vars_self_dens,
                 sizeof(struct pack_vars_self *));
  hipHostMalloc((void **)&pack_vars_self_forc,
                 sizeof(struct pack_vars_self *));
  hipHostMalloc((void **)&pack_vars_self_grad,
                 sizeof(struct pack_vars_self *));

  hipHostMalloc((void **)&pack_vars_pair_dens,
                 sizeof(struct pack_vars_pair *));
  hipHostMalloc((void **)&pack_vars_pair_forc,
                 sizeof(struct pack_vars_pair *));
  hipHostMalloc((void **)&pack_vars_pair_grad,
                 sizeof(struct pack_vars_pair *));
  ///////////////////////////////////////////////////////////////////////////
  /*Find and print GPU name(s)*/
  int devId = 0;  //gpu device name
  struct hipDeviceProp_t prop;
  int nDevices;
  int maxBlocksSM;
  int nSMs;
  /*Get my rank*/
  int mpi_rank = 0;
#ifdef WITH_MPI
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
#endif
  hipGetDeviceCount(&nDevices);
  //A. Nasar: If running on MPI we set code to use one MPI rank per GPU
  //This was found to work very well and simplifies writing slurm scipts
  if (nDevices == 1) hipSetDevice(devId);
#ifdef WITH_MPI
  else {
    hipSetDevice(mpi_rank);
    devId = mpi_rank;
  }
#endif
  //Now tell me some info about my device
  hipGetDeviceProperties(&prop, devId);
  hipDeviceGetAttribute(&maxBlocksSM, hipDeviceAttributeMaxBlocksPerMultiprocessor,
                         devId);
  hipDeviceGetAttribute(&nSMs, hipDeviceAttributeMultiprocessorCount, devId);
  int nPartsPerCell = space->nr_parts / space->tot_cells;

  if (r->cpuid == 0 && mpi_rank == 0) {
    message("%i devices available device id is %i\n", nDevices, devId);
    message("Device : %s\n", prop.name);
    message("nSMs %i max blocks per SM %i maxnBlocks per stream %i\n",
            nSMs, maxBlocksSM, nSMs * maxBlocksSM);
    message("Target nBlocks per kernel is %i\n",
            N_TASKS_BUNDLE_SELF * nPartsPerCell / BLOCK_SIZE);
    message("Target nBlocks per stream is %i\n",
            N_TASKS_PER_PACK_SELF * nPartsPerCell / BLOCK_SIZE);
  }

  hipError_t cu_error;
  size_t free_mem, total_mem;
  hipMemGetInfo(&free_mem, &total_mem);

  message("free mem %lu, total mem %lu", free_mem, total_mem);
  // how many tasks do we want for each launch of GPU kernel
  const int target_n_tasks = sched->pack_size;
  const int target_n_tasks_pair = sched->pack_size_pair;
  pack_vars_self_dens->target_n_tasks = target_n_tasks;
  pack_vars_pair_dens->target_n_tasks = target_n_tasks_pair; //Accounting for one level of recursion
  pack_vars_self_forc->target_n_tasks = target_n_tasks;
  pack_vars_pair_forc->target_n_tasks = target_n_tasks_pair;
  pack_vars_self_grad->target_n_tasks = target_n_tasks;
  pack_vars_pair_grad->target_n_tasks = target_n_tasks_pair;
  // how many tasks we want in each bundle (used for launching kernels in
  // different streams)
  const int bundle_size = N_TASKS_BUNDLE_SELF;
  const int bundle_size_pair = N_TASKS_BUNDLE_PAIR;
  pack_vars_self_dens->bundle_size = bundle_size;
  pack_vars_pair_dens->bundle_size = bundle_size_pair; //Trying to make it so that we have 4 bundles
  pack_vars_self_forc->bundle_size = bundle_size;
  pack_vars_pair_forc->bundle_size = bundle_size_pair;
  pack_vars_self_grad->bundle_size = bundle_size;
  pack_vars_pair_grad->bundle_size = bundle_size_pair;
  // Keep track of first and last particles for each task (particle data is
  // arranged in long arrays containing particles from all the tasks we will
  // work with)
  /* A. N.: Needed for offloading self tasks as we use these to sort through
   *        which parts need to interact with which */
  int2 *task_first_part_f4;
  int2 *task_first_part_f4_f;
  int2 *task_first_part_f4_g;
  int2 *d_task_first_part_f4;
  int2 *d_task_first_part_f4_f;
  int2 *d_task_first_part_f4_g;
  hipHostMalloc((void **)&task_first_part_f4, target_n_tasks * sizeof(int2));
  hipMalloc((void **)&d_task_first_part_f4, target_n_tasks * sizeof(int2));
  hipHostMalloc((void **)&task_first_part_f4_f, target_n_tasks * sizeof(int2));
  hipMalloc((void **)&d_task_first_part_f4_f, target_n_tasks * sizeof(int2));
  hipHostMalloc((void **)&task_first_part_f4_g, target_n_tasks * sizeof(int2));
  hipMalloc((void **)&d_task_first_part_f4_g, target_n_tasks * sizeof(int2));

  /*A. N.: Needed but only for small part in launch functions. Might
           be useful for recursion on the GPU so keep for now     */
  int4 *fparti_fpartj_lparti_lpartj_dens;
  int4 *fparti_fpartj_lparti_lpartj_forc;
  int4 *fparti_fpartj_lparti_lpartj_grad;
  hipHostMalloc((void **)&fparti_fpartj_lparti_lpartj_dens,
		  target_n_tasks * 16 * sizeof(int4));
  hipHostMalloc((void **)&fparti_fpartj_lparti_lpartj_forc,
		  target_n_tasks * 16 * sizeof(int4));
  hipHostMalloc((void **)&fparti_fpartj_lparti_lpartj_grad,
		  target_n_tasks * 16 * sizeof(int4));

  /* nBundles is the number of task bundles each
  thread has ==> Used to loop through bundles */
  int nBundles = (target_n_tasks + bundle_size - 1) / bundle_size;
  int nBundles_pair =
      (target_n_tasks_pair + bundle_size_pair - 1) / bundle_size_pair;

  if (r->cpuid == 0) {
    fprintf(stderr, "engine_rank %i cpuid %i nBundles/nStreams %i\n",
            engine_rank, r->cpuid, nBundles);
    fprintf(stderr, "nBundles/nStreams Pair %i\n", nBundles_pair);
  }

  pack_vars_self_dens->nBundles = nBundles;
  pack_vars_pair_dens->nBundles = nBundles_pair;
  pack_vars_self_forc->nBundles = nBundles;
  pack_vars_pair_forc->nBundles = nBundles_pair;
  pack_vars_self_grad->nBundles = nBundles;
  pack_vars_pair_grad->nBundles = nBundles_pair;

  // first part and last part are the first and last particle ids (locally
  // within this thread). A. Nasar: All these are used in GPU offload setup

  hipHostMalloc((void **)&pack_vars_self_dens->bundle_first_part,
                 nBundles * sizeof(int));
  hipHostMalloc((void **)&pack_vars_self_dens->bundle_last_part,
                 nBundles * sizeof(int));
  hipHostMalloc((void **)&pack_vars_self_dens->bundle_first_task_list,
                 nBundles * sizeof(int));

  hipHostMalloc((void **)&pack_vars_pair_dens->bundle_first_part,
                 2 * nBundles_pair * sizeof(int));
  hipHostMalloc((void **)&pack_vars_pair_dens->bundle_last_part,
                 2 * nBundles_pair * sizeof(int));
  hipHostMalloc((void **)&pack_vars_pair_dens->bundle_first_task_list,
                 2 * nBundles_pair * sizeof(int));

  hipHostMalloc((void **)&pack_vars_self_forc->bundle_first_part,
                 nBundles * sizeof(int));
  hipHostMalloc((void **)&pack_vars_self_forc->bundle_last_part,
                 nBundles * sizeof(int));
  hipHostMalloc((void **)&pack_vars_self_forc->bundle_first_task_list,
                 nBundles * sizeof(int));

  hipHostMalloc((void **)&pack_vars_pair_forc->bundle_first_part,
                 2 * nBundles_pair * sizeof(int));
  hipHostMalloc((void **)&pack_vars_pair_forc->bundle_last_part,
                 2 * nBundles_pair * sizeof(int));
  hipHostMalloc((void **)&pack_vars_pair_forc->bundle_first_task_list,
                 2 * nBundles_pair * sizeof(int));

  hipHostMalloc((void **)&pack_vars_self_grad->bundle_first_part,
                 nBundles * sizeof(int));
  hipHostMalloc((void **)&pack_vars_self_grad->bundle_last_part,
                 nBundles * sizeof(int));
  hipHostMalloc((void **)&pack_vars_self_grad->bundle_first_task_list,
                 nBundles * sizeof(int));

  hipHostMalloc((void **)&pack_vars_pair_grad->bundle_first_part,
                 2 * nBundles_pair * sizeof(int));
  hipHostMalloc((void **)&pack_vars_pair_grad->bundle_last_part,
                 2 * nBundles_pair * sizeof(int));
  hipHostMalloc((void **)&pack_vars_pair_grad->bundle_first_task_list,
                 2 * nBundles_pair * sizeof(int));

  /*Create streams so that we can off-load different batches of work in
   * different streams and get some con-CURRENCY! Events used to maximise
   * asynchrony further*/

  hipStream_t stream[nBundles];
  hipStream_t stream_pairs[nBundles_pair];

  hipEvent_t self_end[nBundles];
  for (int i = 0; i < nBundles; i++) hipEventCreate(&self_end[i]);
  hipEvent_t self_end_g[nBundles];
  for (int i = 0; i < nBundles; i++) hipEventCreate(&self_end_g[i]);
  hipEvent_t self_end_f[nBundles];
  for (int i = 0; i < nBundles; i++) hipEventCreate(&self_end_f[i]);

  hipEvent_t pair_end[nBundles_pair];
  for (int i = 0; i < nBundles_pair; i++) hipEventCreate(&pair_end[i]);
  hipEvent_t pair_end_g[nBundles_pair];
  for (int i = 0; i < nBundles_pair; i++) hipEventCreate(&pair_end_g[i]);
  hipEvent_t pair_end_f[nBundles_pair];
  for (int i = 0; i < nBundles_pair; i++) hipEventCreate(&pair_end_f[i]);

  int tasksperbundle = (target_n_tasks + nBundles - 1) / nBundles;
  int tasksperbundle_pair =
      (target_n_tasks_pair + nBundles_pair - 1) / nBundles_pair;

  pack_vars_self_dens->tasksperbundle = tasksperbundle;
  pack_vars_pair_dens->tasksperbundle = tasksperbundle_pair;
  pack_vars_self_forc->tasksperbundle = tasksperbundle;
  pack_vars_pair_forc->tasksperbundle = tasksperbundle_pair;
  pack_vars_self_grad->tasksperbundle = tasksperbundle;
  pack_vars_pair_grad->tasksperbundle = tasksperbundle_pair;

  for (int i = 0; i < nBundles; ++i)
    hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
  for (int i = 0; i < nBundles_pair; ++i)
    hipStreamCreateWithFlags(&stream_pairs[i], hipStreamNonBlocking);

  pack_vars_self_dens->count_parts = 0;
  pack_vars_pair_dens->count_parts = 0;
  pack_vars_self_forc->count_parts = 0;
  pack_vars_pair_forc->count_parts = 0;
  pack_vars_self_grad->count_parts = 0;
  pack_vars_pair_grad->count_parts = 0;

  /*Estimate how many particles to pack for GPU for each GPU launch
   * instruction*/
  int nr_nodes = 1, res = 0;
#ifdef WITH_MPI
  if ((res = MPI_Comm_size(MPI_COMM_WORLD, &nr_nodes)) != MPI_SUCCESS)
    error("MPI_Comm_size failed with error %i.", res);
#endif
  int parts_per_top_level_cell =
      space->nr_local_cells_with_particles /
      space->nr_parts; /*A. Nasar: What I think is a good approximation for
                                   average N particles in each top level cell*/
  float eta_neighbours = e->s->eta_neighbours;
  int np_per_cell = ceil(2.0 * eta_neighbours);
  np_per_cell *= np_per_cell * np_per_cell;
  /*A. Nasar: Increase parts per recursed task-level cell by buffer to
    ensure we allocate enough memory*/
  int buff = ceil(0.5 * np_per_cell);
  /*A. Nasar: Multiplication by 2 is also to ensure we do not over-run
   *  the allocated memory on buffers and GPU. This can happen if calculated h
   * is larger than cell width and splitting makes bigger than target cells*/
  int count_max_parts_tmp = 64 * 8 * target_n_tasks * (np_per_cell + buff);

  pack_vars_self_dens->count_max_parts = count_max_parts_tmp;
  pack_vars_pair_dens->count_max_parts = count_max_parts_tmp;
  pack_vars_self_forc->count_max_parts = count_max_parts_tmp;
  pack_vars_pair_forc->count_max_parts = count_max_parts_tmp;
  pack_vars_self_grad->count_max_parts = count_max_parts_tmp;
  pack_vars_pair_grad->count_max_parts = count_max_parts_tmp;

  /*Declare Buffer and GPU particle arrays*/
  struct part_aos_f4_send *parts_aos_f4_send;
  struct part_aos_f4_recv *parts_aos_f4_recv;

  struct part_aos_f4_f_send *parts_aos_forc_f4_send;
  struct part_aos_f4_f_recv *parts_aos_forc_f4_recv;

  struct part_aos_f4_g_send *parts_aos_grad_f4_send;
  struct part_aos_f4_g_recv *parts_aos_grad_f4_recv;

  struct part_aos_f4_send *d_parts_aos_f4_send;
  struct part_aos_f4_recv *d_parts_aos_f4_recv;

  struct part_aos_f4_f_send *d_parts_aos_forc_f4_send;
  struct part_aos_f4_f_recv *d_parts_aos_forc_f4_recv;

  struct part_aos_f4_g_send *d_parts_aos_grad_f4_send;
  struct part_aos_f4_g_recv *d_parts_aos_grad_f4_recv;

  struct part_aos_f4_send *parts_aos_pair_f4_send;
  struct part_aos_f4_recv *parts_aos_pair_f4_recv;

  struct part_aos_f4_send *d_parts_aos_pair_f4_send;
  struct part_aos_f4_recv *d_parts_aos_pair_f4_recv;

  struct part_aos_f4_f_send *parts_aos_pair_f4_f_send;
  struct part_aos_f4_f_recv *parts_aos_pair_f4_f_recv;

  struct part_aos_f4_f_send *d_parts_aos_pair_f4_f_send;
  struct part_aos_f4_f_recv *d_parts_aos_pair_f4_f_recv;

  struct part_aos_f4_g_send *parts_aos_pair_f4_g_send;
  struct part_aos_f4_g_recv *parts_aos_pair_f4_g_recv;

  struct part_aos_f4_g_send *d_parts_aos_pair_f4_g_send;
  struct part_aos_f4_g_recv *d_parts_aos_pair_f4_g_recv;

  /*Now allocate memory for Buffer and GPU particle arrays*/
  hipMalloc((void **)&d_parts_aos_f4_send,
             count_max_parts_tmp * sizeof(struct part_aos_f4_send));
  hipMalloc((void **)&d_parts_aos_f4_recv,
             count_max_parts_tmp * sizeof(struct part_aos_f4_recv));

  hipMalloc((void **)&d_parts_aos_forc_f4_send,
             count_max_parts_tmp * sizeof(struct part_aos_f4_f_send));
  hipMalloc((void **)&d_parts_aos_forc_f4_recv,
             count_max_parts_tmp * sizeof(struct part_aos_f4_f_recv));

  hipMalloc((void **)&d_parts_aos_grad_f4_send,
             count_max_parts_tmp * sizeof(struct part_aos_f4_g_send));
  hipMalloc((void **)&d_parts_aos_grad_f4_recv,
             count_max_parts_tmp * sizeof(struct part_aos_f4_g_recv));

  hipHostMalloc((void **)&parts_aos_f4_send,
                 count_max_parts_tmp * sizeof(struct part_aos_f4_send));
  hipHostMalloc((void **)&parts_aos_f4_recv,
                 count_max_parts_tmp * sizeof(struct part_aos_f4_recv));

  hipHostMalloc((void **)&parts_aos_forc_f4_send,
                 count_max_parts_tmp * sizeof(struct part_aos_f4_f_send));
  hipHostMalloc((void **)&parts_aos_forc_f4_recv,
                 count_max_parts_tmp * sizeof(struct part_aos_f4_f_recv));

  hipHostMalloc((void **)&parts_aos_grad_f4_send,
                 count_max_parts_tmp * sizeof(struct part_aos_f4_g_send));
  hipHostMalloc((void **)&parts_aos_grad_f4_recv,
                 count_max_parts_tmp * sizeof(struct part_aos_f4_g_recv));

  hipMalloc((void **)&d_parts_aos_pair_f4_send,
             2 * count_max_parts_tmp * sizeof(struct part_aos_f4_send));
  hipMalloc((void **)&d_parts_aos_pair_f4_recv,
             2 * count_max_parts_tmp * sizeof(struct part_aos_f4_recv));

  hipMalloc((void **)&d_parts_aos_pair_f4_f_send,
             2 * count_max_parts_tmp * sizeof(struct part_aos_f4_f_send));
  hipMalloc((void **)&d_parts_aos_pair_f4_f_recv,
             2 * count_max_parts_tmp * sizeof(struct part_aos_f4_f_recv));

  hipMalloc((void **)&d_parts_aos_pair_f4_g_send,
             2 * count_max_parts_tmp * sizeof(struct part_aos_f4_g_send));
  hipMalloc((void **)&d_parts_aos_pair_f4_g_recv,
             2 * count_max_parts_tmp * sizeof(struct part_aos_f4_g_recv));

  hipHostMalloc((void **)&parts_aos_pair_f4_send,
                 2 * count_max_parts_tmp * sizeof(struct part_aos_f4_send));
  hipHostMalloc((void **)&parts_aos_pair_f4_recv,
                 2 * count_max_parts_tmp * sizeof(struct part_aos_f4_recv));

  hipHostMalloc((void **)&parts_aos_pair_f4_g_send,
                 2 * count_max_parts_tmp * sizeof(struct part_aos_f4_g_send));
  hipHostMalloc((void **)&parts_aos_pair_f4_g_recv,
                 2 * count_max_parts_tmp * sizeof(struct part_aos_f4_g_recv));

  hipHostMalloc((void **)&parts_aos_pair_f4_f_send,
                 2 * count_max_parts_tmp * sizeof(struct part_aos_f4_f_send));
  hipHostMalloc((void **)&parts_aos_pair_f4_f_recv,
                 2 * count_max_parts_tmp * sizeof(struct part_aos_f4_f_recv));

  /*Declare some global variables*/
  float d_a = e->cosmology->a;
  float d_H = e->cosmology->H;
  int step = 0;

  // a list of the cells and tasks the GPU will work on
  pack_vars_self_dens->task_list =
      (struct task **)calloc(target_n_tasks, sizeof(struct task *));
  pack_vars_self_dens->cell_list =
      (struct cell **)calloc(target_n_tasks, sizeof(struct cell *));
  //A. Nasar: The pair version of these is no longer required
  /*Allocate memory for n_leaves_max task pointers per top level task*/
  pack_vars_pair_dens->ci_list =
      (struct cell **)calloc(target_n_tasks_pair, sizeof(struct cell *));
  pack_vars_pair_dens->cj_list =
      (struct cell **)calloc(target_n_tasks_pair, sizeof(struct cell *));

  pack_vars_self_forc->task_list =
      (struct task **)calloc(target_n_tasks, sizeof(struct task *));
  pack_vars_self_forc->cell_list =
      (struct cell **)calloc(target_n_tasks, sizeof(struct cell *));

  pack_vars_pair_forc->ci_list =
      (struct cell **)calloc(target_n_tasks_pair, sizeof(struct cell *));
  pack_vars_pair_forc->cj_list =
      (struct cell **)calloc(target_n_tasks_pair, sizeof(struct cell *));

  pack_vars_self_grad->task_list =
      (struct task **)calloc(target_n_tasks, sizeof(struct task *));
  pack_vars_self_grad->cell_list =
      (struct cell **)calloc(target_n_tasks, sizeof(struct cell *));

  pack_vars_pair_grad->ci_list =
      (struct cell **)calloc(target_n_tasks_pair, sizeof(struct cell *));
  pack_vars_pair_grad->cj_list =
      (struct cell **)calloc(target_n_tasks_pair, sizeof(struct cell *));

  pack_vars_pair_dens->top_task_list =
      (struct task **)calloc(target_n_tasks_pair, sizeof(struct task *));
  pack_vars_pair_grad->top_task_list =
      (struct task **)calloc(target_n_tasks_pair, sizeof(struct task *));
  pack_vars_pair_forc->top_task_list =
      (struct task **)calloc(target_n_tasks_pair, sizeof(struct task *));

  int n_leaves_max = 128;
  //A. Nasar: Over-setimate better than under-estimate
  /*Allocate target_n_tasks for top level tasks. This is a 2D array with length target_n_tasks and width n_leaves_max*/
  int max_length = 2 * target_n_tasks_pair * n_leaves_max;
  struct cell **ci_dd = malloc(max_length * sizeof(struct cell *));
  struct cell **cj_dd = malloc(max_length * sizeof(struct cell *));
  struct cell **ci_dg = malloc(max_length * sizeof(struct cell *));
  struct cell **cj_dg = malloc(max_length * sizeof(struct cell *));
  struct cell **ci_df = malloc(max_length * sizeof(struct cell *));
  struct cell **cj_df = malloc(max_length * sizeof(struct cell *));
  int **first_and_last_daughters_d;
  int **first_and_last_daughters_g;
  int **first_and_last_daughters_f;
  int launch_count = 0;

  for(int i = 0; i < max_length; i++){
	  ci_dd[i] = malloc(sizeof(struct cell *));
	  cj_dd[i] = malloc(sizeof(struct cell *));
	  ci_dg[i] = malloc(sizeof(struct cell *));
	  cj_dg[i] = malloc(sizeof(struct cell *));
	  ci_df[i] = malloc(sizeof(struct cell *));
	  cj_df[i] = malloc(sizeof(struct cell *));
  }

  struct cell **ci_top_d = malloc(2 * target_n_tasks_pair * sizeof(struct cell *));
  struct cell **cj_top_d = malloc(2 * target_n_tasks_pair * sizeof(struct cell *));
  struct cell **ci_top_g = malloc(2 * target_n_tasks_pair * sizeof(struct cell *));
  struct cell **cj_top_g = malloc(2 * target_n_tasks_pair * sizeof(struct cell *));
  struct cell **ci_top_f = malloc(2 * target_n_tasks_pair * sizeof(struct cell *));
  struct cell **cj_top_f = malloc(2 * target_n_tasks_pair * sizeof(struct cell *));
  for(int i = 0; i < 2 * target_n_tasks_pair; i++){
	  ci_top_d[i] = malloc(sizeof(struct cell *));
	  cj_top_d[i] = malloc(sizeof(struct cell *));
	  ci_top_g[i] = malloc(sizeof(struct cell *));
	  cj_top_g[i] = malloc(sizeof(struct cell *));
	  ci_top_f[i] = malloc(sizeof(struct cell *));
	  cj_top_f[i] = malloc(sizeof(struct cell *));
  }
  //change above declaration to the assignment below
  first_and_last_daughters_d = malloc(target_n_tasks_pair * n_leaves_max * sizeof(int *));
  first_and_last_daughters_g = malloc(target_n_tasks_pair * n_leaves_max * sizeof(int *));
  first_and_last_daughters_f = malloc(target_n_tasks_pair * n_leaves_max * sizeof(int *));
  for (int i = 0; i < target_n_tasks_pair * n_leaves_max; i++){
	  first_and_last_daughters_d[i] = malloc(2 * sizeof(int));
	  first_and_last_daughters_g[i] = malloc(2 * sizeof(int));
	  first_and_last_daughters_f[i] = malloc(2 * sizeof(int));
  }

  // number of density self tasks executed
  int tasks_done_cpu = 0;
  int tasks_done_gpu = 0;
  int tasks_done_gpu_inc = 0;

  /* Main loop. */
  while (1) {
    /*Stuff for debugging*/
    int n_full_d_bundles = 0, n_full_g_bundles = 0, n_full_f_bundles = 0;
    int n_full_p_d_bundles = 0, n_full_p_g_bundles = 0, n_full_p_f_bundles = 0;
    int n_partial_d_bundles = 0, n_partial_g_bundles = 0,
        n_partial_f_bundles = 0;
    int n_partial_p_d_bundles = 0, n_partial_p_g_bundles = 0,
        n_partial_p_f_bundles = 0;
    int output = 0;
    int packed_self = 0;
    int packed_pair = 0;
    int packed_self_f = 0;
    int packed_pair_f = 0;
    int packed_self_g = 0;
    int packed_pair_g = 0;
    int last_launched = 0;
    int density = 0;
    int density_sub = 0;
    int unpacked = 0;
    int unpacked_f = 0;
    int unpacked_g = 0;
    int unpacked_pair = 0;
    int unpacked_pair_f = 0;
    int unpacked_pair_g = 0;
    int ghost_in = 0;
    int cpu_self = 0;
    int cpu_self_f = 0;
    int cpu_self_g = 0;
    int cpu_pair = 0;
    int cpu_pair_f = 0;
    int cpu_pair_g = 0;
    int n_leafs_total = 0;
    //	Initialise timers to zero
    double time_for_density_cpu = 0.0;
    double time_for_density_cpu_pair = 0.0;
    double time_for_cpu_g = 0.0;
    double time_for_cpu_pair_g = 0.0;
    double time_for_cpu_f = 0.0;
    double time_for_cpu_pair_f = 0.0;
    double time_for_density_cpu_sub = 0.0;
    double time_for_density_gpu = 0.0;
    double time_for_density_gpu_pair = 0.0;
    double time_for_gpu_f = 0.0;
    double time_for_gpu_pair_f = 0.0;
    double time_for_gpu_g = 0.0;
    double time_for_gpu_pair_g = 0.0;
    double unpack_time_self_g = 0.0;
    double unpack_time_self_f = 0.0;
    double unpack_time_self = 0.0;
    double time_for_gpu_pair = 0.0;
    int nr_cells = space->nr_cells;
    int tops_packed_in_step = 0;
    int n_tops_reset = 0;
    /* Wait at the barrier. */
    engine_barrier(e);
    // Initialise packing counters
    pack_vars_self_dens->tasks_packed = 0;
    pack_vars_pair_dens->tasks_packed = 0;
    pack_vars_self_dens->count_parts = 0;
    pack_vars_pair_dens->count_parts = 0;
    pack_vars_pair_dens->task_locked = 0;
    pack_vars_pair_dens->top_tasks_packed = 0;
    // Initialise packing counters
    pack_vars_self_forc->tasks_packed = 0;
    pack_vars_pair_forc->tasks_packed = 0;
    pack_vars_self_forc->count_parts = 0;
    pack_vars_pair_forc->count_parts = 0;
    // Initialise packing counters
    pack_vars_self_grad->tasks_packed = 0;
    pack_vars_pair_grad->tasks_packed = 0;
    pack_vars_self_grad->count_parts = 0;
    pack_vars_pair_grad->count_parts = 0;

    int total_tasks_packed_this_time_pair = 0;
    double packing_time = 0.0;
    double packing_time_f = 0.0;
    double packing_time_g = 0.0;
    double unpacking_time = 0.0;
    double unpacking_time_f = 0.0;
    double unpacking_time_g = 0.0;
    double packing_time_pair = 0.0;
    double packing_time_pair_f = 0.0;
    double packing_time_pair_g = 0.0;
    double unpacking_time_pair = 0.0;
    double unpacking_time_pair_f = 0.0;
    double unpacking_time_pair_g = 0.0;
    double time_for_copy_to_struct = 0.0;
    double tot_time_for_hard_memcpys = 0.0;
    /* Can we go home yet? */
    if (e->step_props & engine_step_prop_done) break;

    /* Re-set the pointer to the previous task, as there is none. */
    struct task *t = NULL;
    struct task *prev = NULL;
    /*Some bits for output in case of debug*/

    //    if (step == 0) hipProfilerStart();
    step++;

    sched->nr_packs_self_dens_done = 0;
    sched->nr_packs_pair_dens_done = 0;
    sched->nr_packs_self_forc_done = 0;
    sched->nr_packs_pair_forc_done = 0;
    sched->nr_packs_self_grad_done = 0;
    sched->nr_packs_pair_grad_done = 0;

    pack_vars_pair_dens->n_daughters_total = 0;
    pack_vars_pair_grad->n_daughters_total = 0;
    pack_vars_pair_forc->n_daughters_total = 0;

    int n_cells_d = 0;
    int n_cells_g = 0;
    int n_cells_f = 0;
    int n_cells_p_d = 0;
    int n_cells_p_g = 0;
    int n_cells_p_f = 0;
    int n_w_prts_gtr_target_d = 0;
    int n_w_prts_gtr_target_g = 0;
    int n_w_prts_gtr_target_f = 0;
    int n_w_prts_gtr_target_p_d = 0;
    int n_w_prts_gtr_target_p_g = 0;
    int n_w_prts_gtr_target_p_f = 0;
    int leftover_launch_count = 0;
    int g100 = 0;
    int l100 = 0;
    int maxcount = 0;
    /* Loop while there are tasks... */
    tasks_done_gpu_inc = 0;
    ticks hang_time = getticks();
    struct task * ttop_prev;
    while (1) {
      // A. Nasar: Get qid for re-use later
      int qid = r->qid;
      /* If there's no old task, try to get a new one. */
      if (t == NULL) {
        /* Get the task. */
        TIMER_TIC
        t = scheduler_gettask(sched, qid, prev);
        TIMER_TOC(timer_gettask);
        /* Did I get anything? */
        if (t == NULL) break;
      }
      /* Get the cells. */
      struct cell *ci = t->ci;
      struct cell *cj = t->cj;

      struct task * ttop = t;

      if (ci == NULL && (t->subtype != task_subtype_gpu_unpack_d
    		  && t->subtype != task_subtype_gpu_unpack_g
			  && t->subtype != task_subtype_gpu_unpack_f)) error("This cannot be");

#ifdef SWIFT_DEBUG_TASKS
      /* Mark the thread we run on */
      t->rid = r->cpuid;

      /* And recover the pair direction */
      if (t->type == task_type_pair) {
        struct cell *ci_temp = ci;
        struct cell *cj_temp = cj;
        double shift[3];
        if (t->subtype != task_subtype_gpu_unpack_d &&
            t->subtype != task_subtype_gpu_unpack_g &&
            t->subtype != task_subtype_gpu_unpack_f)
          t->sid = space_getsid_and_swap_cells(e->s, &ci_temp, &cj_temp, shift);
      } else {
        t->sid = -1;
      }
#endif

#ifdef SWIFT_DEBUG_CHECKS
      /* Check that we haven't scheduled an inactive task */
      t->ti_run = e->ti_current;
      /* Store the task that will be running (for debugging only) */
      r->t = t;
#endif

      const ticks task_beg = getticks();
      /* Different types of tasks... */
      switch (t->type) {

        case task_type_self:
          if (t->subtype == task_subtype_grav)
            runner_doself_recursive_grav(r, ci, 1);
          else if (t->subtype == task_subtype_external_grav)
            runner_do_grav_external(r, ci, 1);
          else if (t->subtype == task_subtype_gpu_unpack_d) {
            unpacked++;
          } else if (t->subtype == task_subtype_gpu_unpack_g) {
            unpacked_g++;
          } else if (t->subtype == task_subtype_gpu_unpack_f) {
            unpacked_f++;
          } else if (t->subtype == task_subtype_density) {
#ifndef GPUOFFLOAD_DENSITY
            runner_dosub_self1_density(r, ci, /*below_h_max=*/0, 1);
#endif
            /* GPU WORK */
          } else if (t->subtype == task_subtype_gpu_pack_d) {
            packed_self++;
#ifdef GPUOFFLOAD_DENSITY
            runner_doself1_pack_f4(r, sched, pack_vars_self_dens, ci, t,
                                       parts_aos_f4_send, task_first_part_f4);
            /* No pack tasks left in queue, flag that we want to run */
            int launch_leftovers = pack_vars_self_dens->launch_leftovers;
            /*Packed enough tasks. Let's go*/
            int launch = pack_vars_self_dens->launch;
            /* Do we have enough stuff to run the GPU ? */
            if (launch || launch_leftovers) {
              /*Launch GPU tasks*/
              runner_doself1_launch_f4(
                  r, sched, pack_vars_self_dens, ci, t, parts_aos_f4_send,
                  parts_aos_f4_recv, d_parts_aos_f4_send, d_parts_aos_f4_recv,
                  stream, d_a, d_H, e, &packing_time, &time_for_density_gpu,
                  &unpack_time_self, devId,
                  task_first_part_f4, d_task_first_part_f4, self_end);
            } /*End of GPU work Self*/
#endif
          } /* self / pack */
          else if (t->subtype == task_subtype_gpu_pack_g) {
#ifdef GPUOFFLOAD_GRADIENT
            packing_time_g += runner_doself1_pack_f4_g(
                r, sched, pack_vars_self_grad, ci, t, parts_aos_grad_f4_send,
                task_first_part_f4_g);
            /* No pack tasks left in queue, flag that we want to run */
            int launch_leftovers = pack_vars_self_grad->launch_leftovers;
            /*Packed enough tasks let's go*/
            int launch = pack_vars_self_grad->launch;
            /* Do we have enough stuff to run the GPU ? */
            if (launch || launch_leftovers) {
              /*Launch GPU tasks*/
              runner_doself1_launch_f4_g(
                  r, sched, pack_vars_self_grad, ci, t, parts_aos_grad_f4_send,
                  parts_aos_grad_f4_recv, d_parts_aos_grad_f4_send,
                  d_parts_aos_grad_f4_recv, stream, d_a, d_H, e,
                  &packing_time_g, &time_for_gpu_g, task_first_part_f4_g,
                  d_task_first_part_f4_g, self_end_g, &unpack_time_self_g);
            } /*End of GPU work Self*/
#endif  // GPUGRADSELF
          } else if (t->subtype == task_subtype_gpu_pack_f) {
#ifdef GPUOFFLOAD_FORCE
            packing_time_f += runner_doself1_pack_f4_f(
                r, sched, pack_vars_self_forc, ci, t, parts_aos_forc_f4_send,
                task_first_part_f4_f);
            /* No pack tasks left in queue, flag that we want to run */
            int launch_leftovers = pack_vars_self_forc->launch_leftovers;
            /*Packed enough tasks let's go*/
            int launch = pack_vars_self_forc->launch;
            /* Do we have enough stuff to run the GPU ? */
            if (launch || launch_leftovers) {
              /*Launch GPU tasks*/
              runner_doself1_launch_f4_f(
                  r, sched, pack_vars_self_forc, ci, t, parts_aos_forc_f4_send,
                  parts_aos_forc_f4_recv, d_parts_aos_forc_f4_send,
                  d_parts_aos_forc_f4_recv, stream, d_a, d_H, e,
                  &packing_time_f, &time_for_gpu_f, task_first_part_f4_f,
                  d_task_first_part_f4_f, self_end_f, &unpack_time_self_f);
            } /*End of GPU work Self*/
#endif
          }
#ifdef EXTRA_HYDRO_LOOP
          else if (t->subtype == task_subtype_gradient) {
#ifndef GPUOFFLOAD_GRADIENT
#ifdef EXTRA_HYDRO_LOOP_TYPE2
            runner_dosub_self2_gradient(r, ci, /*below_h_max=*/0, 1);
#else
            runner_dosub_self1_gradient(r, ci, /*below_h_max=*/0, 1);
#endif 
#endif
          }
#endif
          else if (t->subtype == task_subtype_force) {
            cpu_self_f++;
#ifndef GPUOFFLOAD_FORCE
            runner_dosub_self2_force(r, ci, /*below_h_max=*/0, 1);
#endif
          } else if (t->subtype == task_subtype_limiter)
            runner_dosub_self1_limiter(r, ci, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_stars_density)
            runner_dosub_self_stars_density(r, ci, /*below_h_max=*/0, 1);
#ifdef EXTRA_STAR_LOOPS
          else if (t->subtype == task_subtype_stars_prep1)
            runner_dosub_self_stars_prep1(r, ci, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_stars_prep2)
            runner_dosub_self_stars_prep2(r, ci, /*below_h_max=*/0, 1);
#endif
          else if (t->subtype == task_subtype_stars_feedback)
            runner_dosub_self_stars_feedback(r, ci, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_bh_density)
            runner_dosub_self_bh_density(r, ci, 1);
          else if (t->subtype == task_subtype_bh_swallow)
            runner_dosub_self_bh_swallow(r, ci, 1);
          else if (t->subtype == task_subtype_do_gas_swallow)
            runner_do_gas_swallow_self(r, ci, 1);
          else if (t->subtype == task_subtype_do_bh_swallow)
            runner_do_bh_swallow_self(r, ci, 1);
          else if (t->subtype == task_subtype_bh_feedback)
            runner_dosub_self_bh_feedback(r, ci, 1);
          else if (t->subtype == task_subtype_rt_gradient)
            runner_dosub_self1_rt_gradient(r, ci, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_rt_transport)
            runner_dosub_self2_rt_transport(r, ci, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_sink_density)
            runner_dosub_self_sinks_density(r, ci, 1);
          else if (t->subtype == task_subtype_sink_swallow)
            runner_dosub_self_sinks_swallow(r, ci, 1);
          else if (t->subtype == task_subtype_sink_do_gas_swallow)
            runner_do_sinks_gas_swallow_self(r, ci, 1);
          else if (t->subtype == task_subtype_sink_do_sink_swallow)
            runner_do_sinks_sink_swallow_self(r, ci, 1);
          else
            error("Unknown/invalid task subtype (%s/%s).",
                  taskID_names[t->type], subtaskID_names[t->subtype]);
          break;

        case task_type_pair:
          if (t->subtype == task_subtype_grav)
            runner_dopair_recursive_grav(r, ci, cj, 1);
          else if (t->subtype == task_subtype_density) {
#ifndef GPUOFFLOAD_DENSITY
            runner_dosub_pair1_density(r, ci, cj, /*below_h_max=*/0, 1);
#endif
          }
          /* GPU WORK */
          else if (t->subtype == task_subtype_gpu_pack_d) {
            packed_pair++;
#ifdef GPUOFFLOAD_DENSITY
#ifndef RECURSE
            ticks tic_cpu_pack = getticks();
            /*Pack data and increment counters checking if we should run on the GPU after packing this task*/
            packing_time_pair +=
                runner_dopair1_pack_f4_d(r, sched, pack_vars_pair_dens, ci,
                                         cj, t, parts_aos_pair_f4_send, e,
                                         fparti_fpartj_lparti_lpartj_dens);
            /* No pack tasks left in queue, flag that we want to run */
            int launch_leftovers = pack_vars_pair_dens->launch_leftovers;
            /*Packed enough tasks let's go*/
            int launch = pack_vars_pair_dens->launch;
            /* Do we have enough stuff to run the GPU ? */
            if (launch || launch_leftovers) {
              /*Launch GPU tasks*/
              runner_dopair1_launch_f4_one_memcpy(
                  r, sched, pack_vars_pair_dens, t, parts_aos_pair_f4_send,
                  parts_aos_pair_f4_recv, d_parts_aos_pair_f4_send,
                  d_parts_aos_pair_f4_recv, stream_pairs, d_a, d_H, e,
                  &packing_time_pair, &time_for_gpu_pair,
                  &unpacking_time_pair, fparti_fpartj_lparti_lpartj_dens,
                  pair_end);
              pack_vars_pair_dens->launch_leftovers = 0;
            } /* End of GPU work Pairs */

#else //RECURSE
            /////////////////////W.I.P!!!////////////////////////////////////////////////////////
            /*Call recursion here. This will be a function in runner_doiact_functions_hydro_gpu.h.
            * We are recursing separately to find out how much work we have before offloading*/
            //We need to allocate a list to put cell pointers into for each new task
            int n_expected_tasks = 4096; //A. Nasar: Need to come up with a good estimate for this
            int depth = 0;
            pack_vars_pair_dens->n_daughters_packed_index = pack_vars_pair_dens->n_daughters_total;
            int n_leaves_found = 0;
            runner_recurse_gpu(r, sched, pack_vars_pair_dens, ci, cj, t,
                      parts_aos_pair_f4_send, e, fparti_fpartj_lparti_lpartj_dens, &n_leaves_found, depth, n_expected_tasks, ci_dd, cj_dd, pack_vars_pair_dens->n_daughters_total);

            runner_pack_daughters_and_launch(r, sched, ci, cj, pack_vars_pair_dens,
            	    t, parts_aos_pair_f4_send , parts_aos_pair_f4_recv, d_parts_aos_pair_f4_send,
            	    parts_aos_pair_f4_recv, stream_pairs, d_a, d_H, e, &packing_time_pair, &time_for_gpu_pair,
            	    &unpacking_time_pair, fparti_fpartj_lparti_lpartj_dens,
            	    pair_end, n_leaves_found, ci_dd, cj_dd, first_and_last_daughters_d, ci_top_d, cj_top_d);
        ///////
#endif  //RECURSE
#endif  // GPUOFFLOAD_DENSITY
          } /* pair / pack */
          else if (t->subtype == task_subtype_gpu_pack_g) {
#ifdef GPUOFFLOAD_GRADIENT
#ifndef RECURSE
              ticks tic_cpu_pack = getticks();
              packing_time_pair_g +=
                  runner_dopair1_pack_f4_g(r, sched, pack_vars_pair_grad, ci,
                                           cj, t, parts_aos_pair_f4_g_send, e,
                                           fparti_fpartj_lparti_lpartj_grad);
              t->total_cpu_pack_ticks += getticks() - tic_cpu_pack;
              /* No pack tasks left in queue, flag that we want to run */
              int launch_leftovers = pack_vars_pair_grad->launch_leftovers;
              /*Packed enough tasks, let's go*/
              int launch = pack_vars_pair_grad->launch;
              /* Do we have enough stuff to run the GPU ? */
              if (launch || launch_leftovers) {
                /*Launch GPU tasks*/
                int t_packed = pack_vars_pair_grad->tasks_packed;
                //                signal_sleeping_runners(sched, t, t_packed);
                runner_dopair1_launch_f4_g_one_memcpy(
                    r, sched, pack_vars_pair_grad, t, parts_aos_pair_f4_g_send,
                    parts_aos_pair_f4_g_recv, d_parts_aos_pair_f4_g_send,
                    d_parts_aos_pair_f4_g_recv, stream_pairs, d_a, d_H, e,
                    &packing_time_pair_g, &time_for_gpu_pair_g,
                    &unpacking_time_pair_g, fparti_fpartj_lparti_lpartj_grad,
                    pair_end_g);
              }
              pack_vars_pair_grad->launch_leftovers = 0;
#else
              /////////////////////W.I.P!!!////////////////////////////////////////////////////////
              /*Call recursion here. This will be a function in runner_doiact_functions_hydro_gpu.h.
              * We are recursing separately to find out how much work we have before offloading*/
              //We need to allocate a list to put cell pointers into for each new task
              int n_expected_tasks = 4096; //A. Nasar: Need to come up with a good estimate for this
              int depth = 0;
              pack_vars_pair_grad->n_daughters_packed_index = pack_vars_pair_grad->n_daughters_total;
              int n_leaves_found = 0;
              runner_recurse_gpu(r, sched, pack_vars_pair_grad, ci, cj, t,
                        parts_aos_pair_f4_g_send, e, fparti_fpartj_lparti_lpartj_grad, &n_leaves_found, depth, n_expected_tasks, ci_dg, cj_dg, pack_vars_pair_grad->n_daughters_total);

              runner_pack_daughters_and_launch_g(r, sched, ci, cj, pack_vars_pair_grad,
              	    t, parts_aos_pair_f4_g_send, parts_aos_pair_f4_g_recv, d_parts_aos_pair_f4_g_send,
              	    parts_aos_pair_f4_g_recv, stream_pairs, d_a, d_H, e, &packing_time_pair_g, &time_for_gpu_pair_g,
              	    &unpacking_time_pair_g, fparti_fpartj_lparti_lpartj_grad,
              	    pair_end_g, n_leaves_found, ci_dg, cj_dg, first_and_last_daughters_g, ci_top_g, cj_top_g);
#endif
#endif  // GPUOFFLOAD_GRADIENT
          } else if (t->subtype == task_subtype_gpu_pack_f) {
            packed_pair_f++;
#ifdef GPUOFFLOAD_FORCE
#ifndef RECURSE
              ticks tic_cpu_pack = getticks();
              /*Pack data and increment counters checking if we should run on the GPU after packing this task*/
              packing_time_pair_f +=
                  runner_dopair1_pack_f4_f(r, sched, pack_vars_pair_forc, ci,
                                           cj, t, parts_aos_pair_f4_f_send, e,
                                           fparti_fpartj_lparti_lpartj_forc);
              /* No pack tasks left in queue, flag that we want to run */
              int launch_leftovers = pack_vars_pair_forc->launch_leftovers;
              /*Packed enough tasks let's go*/
              int launch = pack_vars_pair_forc->launch;
              /* Do we have enough stuff to run the GPU ? */
              if (launch || launch_leftovers) {
                /*Launch GPU tasks*/
                int t_packed = pack_vars_pair_forc->tasks_packed;
                //                signal_sleeping_runners(sched, t, t_packed);
                runner_dopair1_launch_f4_f_one_memcpy(
                    r, sched, pack_vars_pair_forc, t, parts_aos_pair_f4_f_send,
                    parts_aos_pair_f4_f_recv, d_parts_aos_pair_f4_f_send,
                    d_parts_aos_pair_f4_f_recv, stream_pairs, d_a, d_H, e,
                    &packing_time_pair_f, &time_for_gpu_pair_f,
                    &unpacking_time_pair_f, fparti_fpartj_lparti_lpartj_forc,
                    pair_end_f);

                pack_vars_pair_forc->launch_leftovers = 0;
              } /* End of GPU work Pairs */
#else
              /////////////////////W.I.P!!!////////////////////////////////////////////////////////
              /*Call recursion here. This will be a function in runner_doiact_functions_hydro_gpu.h.
              * We are recursing separately to find out how much work we have before offloading*/
              //We need to allocate a list to put cell pointers into for each new task
              int n_expected_tasks = 4096; //A. Nasar: Need to come up with a good estimate for this
              int depth = 0;
              pack_vars_pair_forc->n_daughters_packed_index = pack_vars_pair_forc->n_daughters_total;
              int n_leaves_found = 0;
              runner_recurse_gpu(r, sched, pack_vars_pair_forc, ci, cj, t,
                        parts_aos_pair_f4_f_send, e, fparti_fpartj_lparti_lpartj_forc, &n_leaves_found, depth, n_expected_tasks, ci_df, cj_df, pack_vars_pair_forc->n_daughters_total);

              runner_pack_daughters_and_launch_f(r, sched, ci, cj, pack_vars_pair_forc,
                    t, parts_aos_pair_f4_f_send, parts_aos_pair_f4_f_recv, d_parts_aos_pair_f4_f_send,
                    parts_aos_pair_f4_f_recv, stream_pairs, d_a, d_H, e, &packing_time_pair_f, &time_for_gpu_pair_f,
                    &unpacking_time_pair_f, fparti_fpartj_lparti_lpartj_forc,
                    pair_end_f, n_leaves_found, ci_df, cj_df, first_and_last_daughters_f, ci_top_f, cj_top_f);
#endif
#endif  // GPUOFFLOAD_FORCE
          } else if (t->subtype == task_subtype_gpu_unpack_d) {
            unpacked_pair++;
          } else if (t->subtype == task_subtype_gpu_unpack_g) {
            unpacked_pair_g++;
          } else if (t->subtype == task_subtype_gpu_unpack_f) {
            unpacked_pair_f++;
          }

#ifdef EXTRA_HYDRO_LOOP
          else if (t->subtype == task_subtype_gradient) {
            int Do_nothing = 0;
#ifndef GPUOFFLOAD_GRADIENT
#ifdef EXTRA_HYDRO_LOOP_TYPE2
            runner_dosub_pair2_gradient(r, ci, cj, /*below_h_max=*/0, 1);
#else
            runner_dosub_pair1_gradient(r, ci, cj, /*below_h_max=*/0, 1);
#endif
#endif
          }
#endif  // EXTRA_HYDRO_LOOP
          else if (t->subtype == task_subtype_force) {
#ifndef GPUOFFLOAD_FORCE
            runner_dosub_pair2_force(r, ci, cj, /*below_h_max=*/0, 1);
#endif  // GPUOFFLOAD_FORCE
          } else if (t->subtype == task_subtype_limiter)
            runner_dosub_pair1_limiter(r, ci, cj, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_stars_density)
            runner_dosub_pair_stars_density(r, ci, cj, /*below_h_max=*/0, 1);
#ifdef EXTRA_STAR_LOOPS
          else if (t->subtype == task_subtype_stars_prep1)
            runner_dosub_pair_stars_prep1(r, ci, cj, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_stars_prep2)
            runner_dosub_pair_stars_prep2(r, ci, cj, /*below_h_max=*/0, 1);
#endif
          else if (t->subtype == task_subtype_stars_feedback)
            runner_dosub_pair_stars_feedback(r, ci, cj, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_bh_density)
            runner_dosub_pair_bh_density(r, ci, cj, 1);
          else if (t->subtype == task_subtype_bh_swallow)
            runner_dosub_pair_bh_swallow(r, ci, cj, 1);
          else if (t->subtype == task_subtype_do_gas_swallow)
            runner_do_gas_swallow_pair(r, ci, cj, 1);
          else if (t->subtype == task_subtype_do_bh_swallow)
            runner_do_bh_swallow_pair(r, ci, cj, 1);
          else if (t->subtype == task_subtype_bh_feedback)
            runner_dosub_pair_bh_feedback(r, ci, cj, 1);
          else if (t->subtype == task_subtype_rt_gradient)
            runner_dosub_pair1_rt_gradient(r, ci, cj, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_rt_transport)
            runner_dosub_pair2_rt_transport(r, ci, cj, /*below_h_max=*/0, 1);
          else if (t->subtype == task_subtype_sink_density)
            runner_dosub_pair_sinks_density(r, ci, cj, 1);
          else if (t->subtype == task_subtype_sink_swallow)
            runner_dosub_pair_sinks_swallow(r, ci, cj, 1);
          else if (t->subtype == task_subtype_sink_do_gas_swallow)
            runner_do_sinks_gas_swallow_pair(r, ci, cj, 1);
          else if (t->subtype == task_subtype_sink_do_sink_swallow)
            runner_do_sinks_sink_swallow_pair(r, ci, cj, 1);
          else
            error("Unknown/invalid task subtype (%s/%s).",
                  taskID_names[t->type], subtaskID_names[t->subtype]);
          break;

        case task_type_sort:
          /* Cleanup only if any of the indices went stale. */
          runner_do_hydro_sort(
              r, ci, t->flags,
              ci->hydro.dx_max_sort_old > space_maxreldx * ci->dmin,
              /*lock=*/0, cell_get_flag(ci, cell_flag_rt_requests_sort),
              /*clock=*/1);
          /* Reset the sort flags as our work here is done. */
          t->flags = 0;
          break;
        case task_type_rt_sort:
          /* Cleanup only if any of the indices went stale.
           * NOTE: we check whether we reset the sort flags when the
           * recv tasks are running. Cells without an RT recv task
           * don't have rt_sort tasks. */
          runner_do_hydro_sort(
              r, ci, t->flags,
              ci->hydro.dx_max_sort_old > space_maxreldx * ci->dmin,
              /*lock=*/0, /*rt_requests_sorts=*/1, /*clock=*/1);
          /* Reset the sort flags as our work here is done. */
          t->flags = 0;
          break;
        case task_type_stars_sort:
          /* Cleanup only if any of the indices went stale. */
          runner_do_stars_sort(
              r, ci, t->flags,
              ci->stars.dx_max_sort_old > space_maxreldx * ci->dmin, 1);
          /* Reset the sort flags as our work here is done. */
          t->flags = 0;
          break;
        case task_type_init_grav:
          runner_do_init_grav(r, ci, 1);
          break;
        case task_type_ghost:
          runner_do_ghost(r, ci, 1);
          break;
#ifdef EXTRA_HYDRO_LOOP
        case task_type_extra_ghost:
          runner_do_extra_ghost(r, ci, 1);
          break;
#endif
        case task_type_stars_ghost:
          runner_do_stars_ghost(r, ci, 1);
          break;
        case task_type_bh_density_ghost:
          runner_do_black_holes_density_ghost(r, ci, 1);
          break;
        case task_type_bh_swallow_ghost3:
          runner_do_black_holes_swallow_ghost(r, ci, 1);
          break;
        case task_type_sink_density_ghost:
          runner_do_sinks_density_ghost(r, ci, 1);
          break;
        case task_type_drift_part:
          runner_do_drift_part(r, ci, 1);
          break;
        case task_type_drift_spart:
          runner_do_drift_spart(r, ci, 1);
          break;
        case task_type_drift_sink:
          runner_do_drift_sink(r, ci, 1);
          break;
        case task_type_drift_bpart:
          runner_do_drift_bpart(r, ci, 1);
          break;
        case task_type_drift_gpart:
          runner_do_drift_gpart(r, ci, 1);
          break;
        case task_type_kick1:
          runner_do_kick1(r, ci, 1);
          break;
        case task_type_kick2:
          runner_do_kick2(r, ci, 1);
          break;
        case task_type_end_hydro_force:
          runner_do_end_hydro_force(r, ci, 1);
          break;
        case task_type_end_grav_force:
          runner_do_end_grav_force(r, ci, 1);
          break;
        case task_type_csds:
          runner_do_csds(r, ci, 1);
          break;
        case task_type_timestep:
          runner_do_timestep(r, ci, 1);
          break;
        case task_type_timestep_limiter:
          runner_do_limiter(r, ci, 0, 1);
          break;
        case task_type_timestep_sync:
          runner_do_sync(r, ci, 0, 1);
          break;
        case task_type_collect:
          runner_do_timestep_collect(r, ci, 1);
          break;
        case task_type_rt_collect_times:
          runner_do_collect_rt_times(r, ci, 1);
          break;
#ifdef WITH_MPI
        case task_type_send:
          if (t->subtype == task_subtype_tend) {
            free(t->buff);
          } else if (t->subtype == task_subtype_sf_counts) {
            free(t->buff);
          } else if (t->subtype == task_subtype_grav_counts) {
            free(t->buff);
          } else if (t->subtype == task_subtype_part_swallow) {
            free(t->buff);
          } else if (t->subtype == task_subtype_bpart_merger) {
            free(t->buff);
          } else if (t->subtype == task_subtype_limiter) {
            free(t->buff);
          }
          break;
        case task_type_recv:
          if (t->subtype == task_subtype_tend) {
            cell_unpack_end_step(ci, (struct pcell_step *)t->buff);
            free(t->buff);
          } else if (t->subtype == task_subtype_sf_counts) {
            cell_unpack_sf_counts(ci, (struct pcell_sf_stars *)t->buff);
            cell_clear_stars_sort_flags(ci, /*clear_unused_flags=*/0);
            free(t->buff);
          } else if (t->subtype == task_subtype_grav_counts) {
            cell_unpack_grav_counts(ci, (struct pcell_sf_grav *)t->buff);
            free(t->buff);
          } else if (t->subtype == task_subtype_xv) {
            runner_do_recv_part(r, ci, 1, 1);
          } else if (t->subtype == task_subtype_rho) {
            runner_do_recv_part(r, ci, 0, 1);
          } else if (t->subtype == task_subtype_gradient) {
            runner_do_recv_part(r, ci, 0, 1);
          } else if (t->subtype == task_subtype_rt_gradient) {
            runner_do_recv_part(r, ci, 2, 1);
          } else if (t->subtype == task_subtype_rt_transport) {
            runner_do_recv_part(r, ci, -1, 1);
          } else if (t->subtype == task_subtype_part_swallow) {
            cell_unpack_part_swallow(ci,
                                     (struct black_holes_part_data *)t->buff);
            free(t->buff);
          } else if (t->subtype == task_subtype_bpart_merger) {
            cell_unpack_bpart_swallow(ci,
                                      (struct black_holes_bpart_data *)t->buff);
            free(t->buff);
          } else if (t->subtype == task_subtype_limiter) {
            /* Nothing to do here. Unpacking done in a separate task */
          } else if (t->subtype == task_subtype_gpart) {
            runner_do_recv_gpart(r, ci, 1);
          } else if (t->subtype == task_subtype_spart_density) {
            runner_do_recv_spart(r, ci, 1, 1);
          } else if (t->subtype == task_subtype_part_prep1) {
            runner_do_recv_part(r, ci, 0, 1);
          } else if (t->subtype == task_subtype_spart_prep2) {
            runner_do_recv_spart(r, ci, 0, 1);
          } else if (t->subtype == task_subtype_bpart_rho) {
            runner_do_recv_bpart(r, ci, 1, 1);
          } else if (t->subtype == task_subtype_bpart_feedback) {
            runner_do_recv_bpart(r, ci, 0, 1);
          } else {
            error("Unknown/invalid task subtype (%d).", t->subtype);
          }
          break;

        case task_type_pack:
          runner_do_pack_limiter(r, ci, &t->buff, 1);
          task_get_unique_dependent(t)->buff = t->buff;
          break;
        case task_type_unpack:
          runner_do_unpack_limiter(r, ci, t->buff, 1);
          break;
#endif
        case task_type_grav_down:
          runner_do_grav_down(r, t->ci, 1);
          break;
        case task_type_grav_long_range:
          runner_do_grav_long_range(r, t->ci, 1);
          break;
        case task_type_grav_mm:
          runner_dopair_grav_mm_progenies(r, t->flags, t->ci, t->cj);
          break;
        case task_type_cooling:
          runner_do_cooling(r, t->ci, 1);
          break;
        case task_type_star_formation:
          runner_do_star_formation(r, t->ci, 1);
          break;
        case task_type_star_formation_sink:
          runner_do_star_formation_sink(r, t->ci, 1);
          break;
        case task_type_stars_resort:
          runner_do_stars_resort(r, t->ci, 1);
          break;
        case task_type_sink_formation:
          runner_do_sink_formation(r, t->ci);
          break;
        case task_type_fof_self:
          runner_do_fof_search_self(r, t->ci, 1);
          break;
        case task_type_fof_pair:
          runner_do_fof_search_pair(r, t->ci, t->cj, 1);
          break;
        case task_type_fof_attach_self:
          runner_do_fof_attach_self(r, t->ci, 1);
          break;
        case task_type_fof_attach_pair:
          runner_do_fof_attach_pair(r, t->ci, t->cj, 1);
          break;
        case task_type_neutrino_weight:
          runner_do_neutrino_weighting(r, ci, 1);
          break;
        case task_type_rt_ghost1:
          runner_do_rt_ghost1(r, t->ci, 1);
          break;
        case task_type_rt_ghost2:
          runner_do_rt_ghost2(r, t->ci, 1);
          break;
        case task_type_rt_tchem:
          runner_do_rt_tchem(r, t->ci, 1);
          break;
        case task_type_rt_advance_cell_time:
          runner_do_rt_advance_cell_time(r, t->ci, 1);
          break;
        default:
          error("Unknown/invalid task type (%d).", t->type);
      }
      r->active_time += (getticks() - task_beg);

/* Mark that we have run this task on these cells */
#ifdef SWIFT_DEBUG_CHECKS
      if (ci != NULL) {
        ci->tasks_executed[t->type]++;
        ci->subtasks_executed[t->subtype]++;
      }
      if (cj != NULL) {
        cj->tasks_executed[t->type]++;
        cj->subtasks_executed[t->subtype]++;
      }
      /* This runner is not doing a task anymore */
      r->t = NULL;
#endif
      /* We're done with this task, see if we get a next one. */
      prev = t;
      if (t->subtype == task_subtype_gpu_pack_d) {
#ifdef GPUOFFLOAD_DENSITY
        /* Don't enqueue unpacks yet. Just signal the runners */
        t->skip = 1;
        t->toc = getticks();
        t->total_ticks += t->toc - t->tic;
        t = NULL;
#else
        t = scheduler_done(sched, t);
#endif
      }

      else if (t->subtype == task_subtype_gpu_pack_g) {
#ifdef GPUOFFLOAD_GRADIENT
        /* Don't enqueue unpacks yet. Just signal the runners */
        t->skip = 1;
        t->toc = getticks();
        t->total_ticks += t->toc - t->tic;
        t = NULL;
#else
        t = scheduler_done(sched, t);
#endif
      }

      else if (t->subtype == task_subtype_gpu_pack_f) {
#ifdef GPUOFFLOAD_FORCE
        /* Don't enqueue unpacks yet. Just signal the runners */
        t->skip = 1;
        t->toc = getticks();
        t->total_ticks += t->toc - t->tic;
        t = NULL;
#else
        t = scheduler_done(sched, t);
#endif
      }

      else if (t->subtype != task_subtype_gpu_pack_d &&
               t->subtype != task_subtype_gpu_pack_g &&
               t->subtype != task_subtype_gpu_pack_f) {
        t = scheduler_done(sched, t);
      }
    } /* main loop. */

    time_for_density_cpu = 0.0;
    time_for_density_gpu = 0.0;
    time_for_density_cpu_pair = 0.0;
    time_for_density_gpu_pair = 0.0;
    time_for_density_cpu_sub = 0.0;
    tot_time_for_hard_memcpys = 0.0;
    tasks_done_gpu = 0;
    tasks_done_cpu = 0;
    tasks_done_gpu_inc = 0;
    if (ghost_in > 0)
      fprintf(stderr, "total tasks not done on GPU %i is %i\n", r->cpuid,
              ghost_in);
    packed_self = 0;
    packed_pair = 0;
    packed_self_f = 0;
    packed_pair_f = 0;
    packed_self_g = 0;
    packed_pair_g = 0;
    density = 0;
    density_sub = 0;
    unpacked = 0;
    //	if(step == 2)hipProfilerStop();
    //	if(step == 2)exit(0);
    //	  size_t free_byte ;
    //	  size_t total_byte ;
    //	  hipError_t cuda_status = hipMemGetInfo( &free_byte,
    //&total_byte ) ; 	  double free = (double)free_byte; 	  double
    // available = (double)total_byte; 	  double used = (available - free);
    // fprintf(stderr, "Used %f GB GPU memory\n", used/1e9);
    /* Wait at the wait barrier. */
    //    swift_barrier_wait(&e->wait_barrier);
  }
  // Free all data
  //  hipFree(d_tid_p);
  //  hipFree(d_id);
  //  hipFree(d_x_p);
  //  hipFree(d_y_p);
  //  hipFree(d_z_p);
  //  hipFree(d_ux);
  //  hipFree(d_uy);
  //  hipFree(d_uz);
  //  hipFree(d_a_hydrox);
  //  hipFree(d_a_hydroy);
  //  hipFree(d_a_hydroz);
  //  hipFree(d_mass);
  //  hipFree(d_h);
  //  hipFree(d_u);
  //  hipFree(d_u_dt);
  //  hipFree(d_rho);
  //  hipFree(d_SPH_sum);
  //  hipFree(d_locx);
  //  hipFree(d_locy);
  //  hipFree(d_locz);
  //  hipFree(d_widthx);
  //  hipFree(d_widthy);
  //  hipFree(d_widthz);
  //  hipFree(d_h_max);
  //  hipFree(d_count_p);
  //  hipFree(d_wcount);
  //  hipFree(d_wcount_dh);
  //  hipFree(d_rho_dh);
  //  hipFree(d_rot_ux);
  //  hipFree(d_rot_uy);
  //  hipFree(d_rot_uz);
  //  hipFree(d_div_v);
  //  hipFree(d_div_v_previous_step);
  //  hipFree(d_alpha_visc);
  //  hipFree(d_v_sig);
  //  hipFree(d_laplace_u);
  //  hipFree(d_alpha_diff);
  //  hipFree(d_f);
  //  hipFree(d_soundspeed);
  //  hipFree(d_h_dt);
  //  hipFree(d_balsara);
  //  hipFree(d_pressure);
  //  hipFree(d_alpha_visc_max_ngb);
  //  hipFree(d_time_bin);
  //  hipFree(d_wakeup);
  //  hipFree(d_min_ngb_time_bin);
  //  hipFree(d_to_be_synchronized);
  //  hipFree(tid_p);
  //  hipFree(id);
  //  hipFree(mass);
  //  hipFree(h);
  //  hipFree(u);
  //  hipFree(u_dt);
  //  hipFree(rho);
  //  hipFree(SPH_sum);
  //  hipFree(x_p);
  //  hipFree(y_p);
  //  hipFree(z_p);
  //  hipFree(ux);
  //  hipFree(uy);
  //  hipFree(uz);
  //  hipFree(a_hydrox);
  //  hipFree(a_hydroy);
  //  hipFree(a_hydroz);
  //  hipFree(locx);
  //  hipFree(locy);
  //  hipFree(locz);
  //  hipFree(widthx);
  //  hipFree(widthy);
  //  hipFree(widthz);
  //  hipFree(h_max);
  //  hipFree(count_p);
  //  hipFree(wcount);
  //  hipFree(wcount_dh);
  //  hipFree(rho_dh);
  //  hipFree(rot_ux);
  //  hipFree(rot_uy);
  //  hipFree(rot_uz);
  //  hipFree(div_v);
  //  hipFree(div_v_previous_step);
  //  hipFree(alpha_visc);
  //  hipFree(v_sig);
  //  hipFree(laplace_u);
  //  hipFree(alpha_diff);
  //  hipFree(f);
  //  hipFree(soundspeed);
  //  hipFree(h_dt);
  //  hipFree(balsara);
  //  hipFree(pressure);
  //  hipFree(alpha_visc_max_ngb);
  //  hipFree(time_bin);
  //  hipFree(wakeup);
  //  hipFree(min_ngb_time_bin);
  //  hipFree(to_be_synchronized);
  //  hipFree(partid_p);
  //  hipFree(d_task_first_part);
  //  hipFree(d_task_last_part);
  //  hipFree(task_first_part_self_dens);
  //  hipFree(task_last_part_self_dens);
  //  hipFree(task_first_part_pair_ci);
  //  hipFree(task_last_part_pair_ci);
  //  hipFree(task_first_part_pair_cj);
  //  hipFree(task_last_part_pair_cj);
  //  hipFree(d_bundle_first_part_self_dens);
  //  hipFree(d_bundle_last_part_self_dens);
  //  hipFree(bundle_first_part_self_dens);
  //  hipFree(bundle_last_part_self_dens);
  //  hipFree(bundle_first_part_pair_ci);
  //  hipFree(bundle_last_part_pair_ci);
  //  hipFree(bundle_first_part_pair_cj);
  //  hipFree(bundle_last_part_pair_cj);
  //  free(ci_list_self_dens);
  //  free(ci_list_pair);
  //  free(cj_list_pair);

  /* Be kind, rewind. */
  return NULL;
}

ticks runner_get_active_time(const struct runner *restrict r) {
  return r->active_time;
}

void runner_reset_active_time(struct runner *restrict r) { r->active_time = 0; }

#endif  // WITH_CUDA
